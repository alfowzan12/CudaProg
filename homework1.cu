#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ 
void dkernel(int *arr, int N){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	while(id < N)
	{
		arr[id] = 0;
		id += blockDim.x * gridDim.x ;
	}
}

__global__
void dkernel_add(int *arr, int N)
{
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	while (id < N)
	{
		arr[id] += id;
		id += blockDim.x *  gridDim.x;
	}
}

int main() {
	int *gpuArray, *cpuArray;
	//int *cpuArray = new int[32];
	hipMallocManaged(&cpuArray, 32*sizeof(int));
	
	hipMallocManaged(&gpuArray, 32*sizeof(int));
	dkernel<<<1, 32>>>(gpuArray,32);
	hipMemcpy(cpuArray, gpuArray, 32*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < 32 ; i++)
	{
		cout <<"cpuArray["<<i<<"]"<<cpuArray[i]<< endl;
	}

	hipMallocManaged(&gpuArray, 1024*sizeof(int));
	dkernel<<<1, 1024>>>(gpuArray, 1024);
	dkernel_add<<<1, 1024>>>(gpuArray, 1024);
	//cpuArray = new int[1024];
	hipMallocManaged(&cpuArray, 1024*sizeof(int));
	hipMemcpy(cpuArray, gpuArray, 1024*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0 ; i < 1024 ; i++)
	{
		cout << "cpuArray["<<i<<"]"<< cpuArray[i] << endl;
	}
	
	hipMallocManaged(&gpuArray, 8000*sizeof(int));
	dkernel<<<8000/128, 128>>>(gpuArray, 8000);
	dkernel_add<<<8000,128>>>(gpuArray, 8000);
	cpuArray = new int[8000];
	hipMemcpy(cpuArray, gpuArray, 8000*sizeof(int), hipMemcpyDeviceToHost);

	for(int i = 0; i < 8000 ; i++) 
	{
		cout << "cpuArray["<<i<<"]" << cpuArray[i] << endl;
	}

}
